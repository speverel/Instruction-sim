#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "../cudpp/include/cudpp.h"
#include "../cudpp/include/cudpp_config.h"
#include "../cuda/6.5/include/hipblas.h"
#include <hip/hip_runtime.h>
#define WIDTH 64
#define HEIGHT 64
#define MAX_VAL 120

#define CLEANUP(s, v) \
	do { \
		printf("%s\n", s); \
		if (h_Act)			free(h_Act); \
		if (h_Mat)			free(h_Mat); \
		if (h_Out)			free(h_Out); \
		if (handle)			cudppDestroy(handle); \
		if (blasHandle)		hipblasDestroy(blasHandle); \
		if (data)			hipFree(data); \
		if (startAddr)		hipFree(startAddr); \
		if (endAddr)		hipFree(endAddr); \
		if (*lowestAddr)	hipFree(*lowestAddr); \
		if (output)			hipFree(output); \
		if (indices)		hipFree(indices); \
		if (matrix)			hipFree(matrix); \
		if (P)				hipFree(P); \
		if (blasMatrix)		hipFree(blasMatrix); \
		if (blasVector)		hipFree(blasVector); \
		hipDeviceReset(); \
		fflush(stdout); \
	} while (0); \
	return v;

__global__ void setIndices(int* indices)
{
	unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
	indices[tid] = tid;
}

__global__ void findLowest(float** result, float* data, int bias)
{
	unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (bias == 0 || tid < blockDim.x * gridDim.x - 1) // prevents last thread from accessing off the end
	{
		if (data[2 * tid + bias] == 0 && data[2 * tid + 1 + bias] > 0) // this is the zero boundary if true
			*result = data + 2 * tid + 1 + bias;	
	}
}

// launch endAddr(a) - startAddr(a) blocks of height threads each
__global__ void denseSparseMVM(float* W, float* a, int* ind, unsigned height, unsigned width, float* P)
{
	unsigned tid = threadIdx.x;
	unsigned i = blockIdx.x;
	float* pw = W + ind[i] * height;
	P[tid * height + i] = pw[tid] * a[i];
}

__global__ void clearPsums(float* P)
{
	P[blockIdx.x * blockDim.x + threadIdx.x] = 0.0;
}

/*
__global__ void sparseSparseMVM(float* W, float** Wends, int* Windices, float* a, float* aend, int* aindices, unsigned height, float** P)
{
	extern __shared__ int* Pindices;
	unsigned tid = threadIdx.x;
	for (unsigned i = 0; i <= (aend - a); i++)
	{
		int aindex = aindices[i];
		float* colEnd = Wends[aindex];
		float* colBeg = W + aindex * height;
		if (colBeg + tid <= colEnd)
		{
			int* Wcolind = Windices + aindex * height;
			int windex = Wcolind[tid];
			float psum = colBeg[tid] * a[i];
			unsigned pI = Pindices[windex];
			P[windex][pI] = psum;
			Pindices[windex] = pI + 1;
		}
	}
}	
*/
void merge(int low, int mid, int high, float* idata, float* odata, int* iind, int* oind)
{
	int l1, l2, i;
	for (l1 = low, l2 = mid + 1, i = low; l1 <= mid && l2 <= high; i++)
	{   
		if (idata[l1] <= idata[l2])
		{
			oind[i] = iind[l1];
			odata[i] = idata[l1++];
		}
		else
		{
			oind[i] = iind[l2];
			odata[i] = idata[l2++];
		}
	}
	while (l1 <= mid)
	{
		oind[i] = iind[l1];
		odata[i++] = idata[l1++];
	}
	while (l2 <= high)
	{
		oind[i] = iind[l2];
		odata[i++] = idata[l2++];
	}
	for (i = low; i <= high; i++)
	{
		idata[i] = odata[i];
		iind[i] = oind[i];
	}
}

void mergesort(int low, int high, float* idata, float* odata, int* iind, int* oind)
{
	int mid;
	if (low < high)
	{
		mid = (low + high) / 2;
		mergesort(low, mid, idata, odata, iind, oind);
		mergesort(mid + 1, high, idata, odata, iind, oind);
		merge(low, mid, high, idata, odata, iind, oind);
	}
	else
		return;
}

void matMul(float* matrix, float* vector, float* output, int width, int height)
{
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			output[i] += matrix[j * height + i] * vector[j];
		}
	}
}

void printGold(float* matrix, float* vector, int width, int height, float* dataPtr)
{
	float* tempData = (float*)malloc(sizeof(float) * width);
	int* indices = (int*)malloc(sizeof(int) * width);
	int* tempIndices = (int*)malloc(sizeof(int) * width);
	float* mulResult = (float*)malloc(sizeof(float) * height);
	for (int i = 0; i < width; i++)
		indices[i] = i;
	for (int i = 0; i < height; i++)
		mulResult[i] = 0.0;
	matMul(matrix, vector, mulResult, width, height);
	float *startAddr, *endAddr;
	endAddr = dataPtr + width;
	mergesort(0, width - 1, vector, tempData, indices, tempIndices);
	int index = 0;
	while (tempData[index] == 0.0)
		index++;
	startAddr = dataPtr + index;
	printf("CPU results:\n");
	printf("Start pointer: %p End pointer: %p\n", startAddr, endAddr);
	for (int i = startAddr - dataPtr; i < endAddr - dataPtr; i++)
		printf("Input: %f Index: %d\n", tempData[i], tempIndices[i]);
	for (int i = 0; i < height; i++)
		printf("Output: %f\n", mulResult[i]);	
	free(tempData);
	free(indices);
	free(tempIndices);
	free(mulResult);
}

int main()
{
	srand(time(NULL));
	float* h_Act = (float*)malloc(sizeof(float) * WIDTH);
	float* h_Mat = (float*)malloc(sizeof(float) * WIDTH * HEIGHT);
	float* h_Out = (float*)malloc(sizeof(float) * HEIGHT);
	float temp;
	CUDPPHandle handle = 0;
	CUDPPHandle scanplan = 0;
	hipblasStatus_t stat, stat2;
	hipblasHandle_t blasHandle;
	int nnz = 0;
	for (int i = 0; i < WIDTH; i++) // generate random floats, which are rectified
	{
		temp = (float)rand() / (float)(RAND_MAX / MAX_VAL) - MAX_VAL / 2.5;
		if (temp > 0)
		{
			h_Act[i] = temp;
			nnz++;
		}
		else
		{
			h_Act[i] = 0;
		}
	}

	// populate matrix
	for (int i = 0; i < WIDTH; i++)
	{
		for (int j = 0; j < HEIGHT; j++)
		{
			h_Mat[i * HEIGHT + j] = (float)rand() / (float)(RAND_MAX / (MAX_VAL)) - MAX_VAL / 2;
		}
	}
	// allocate memory on GPU, and copy activation vector over
	float *data, *output, *matrix, *startAddr, *endAddr, *P, *blasMatrix, *blasVector;
	int* indices;
	float** foo;
	float*** lowestAddr = &foo;
	hipError_t err1 = hipMallocManaged(&data, sizeof(float) * WIDTH, hipMemAttachGlobal);
	hipError_t err2 = hipMallocManaged(&indices, sizeof(int) * WIDTH, hipMemAttachGlobal);
	hipError_t err3 = hipMallocManaged(&startAddr, sizeof(float), hipMemAttachGlobal);
	hipError_t err4 = hipMalloc(&output, sizeof(float) * HEIGHT);
	hipError_t err5 = hipMalloc(&matrix, sizeof(float) * HEIGHT * WIDTH);
	hipError_t err6 = hipMallocManaged(lowestAddr, sizeof(float**), hipMemAttachGlobal);
	hipError_t err7 = hipMallocManaged(&endAddr, sizeof(float), hipMemAttachGlobal);
	hipError_t err8 = hipMallocManaged(&P, sizeof(float) * HEIGHT * WIDTH);
	hipError_t err9 = hipMalloc(&blasMatrix, sizeof(float) * HEIGHT * WIDTH);
	hipError_t err10 = hipMalloc(&blasVector, sizeof(float) * WIDTH);
	if (err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess || err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess || err8 != hipSuccess || err9 != hipSuccess || err10 != hipSuccess) 
	{
		CLEANUP("Failed to allocate memory on device.", 1);
	}
	err1 = hipMemcpy(data, h_Act, sizeof(float) * WIDTH, hipMemcpyHostToDevice);
	err3 = hipMemcpy(matrix, h_Mat, sizeof(float) * WIDTH * HEIGHT, hipMemcpyHostToDevice);
	if (err1 != hipSuccess || err3 != hipSuccess)
	{
		CLEANUP("Failed to copy memory to device.", 1);
	}

	stat = hipblasCreate(&blasHandle);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		CLEANUP("Failed to create cuBLAS handle.", 1);
	}
	stat = hipblasSetMatrix(HEIGHT, WIDTH, sizeof(float), h_Mat, HEIGHT, blasMatrix, HEIGHT);
	stat2 = hipblasSetVector(WIDTH, sizeof(float), h_Act, 1, blasVector, 1);
	if (stat != HIPBLAS_STATUS_SUCCESS || stat2 != HIPBLAS_STATUS_SUCCESS)
	{
		CLEANUP("Failed to copy BLAS matrix or vector over.", 1);
	}
	printGold(h_Mat, h_Act, WIDTH, HEIGHT, data);

	setIndices<<<4, WIDTH / 4>>>(indices);
	clearPsums<<<HEIGHT, WIDTH>>>(P);

	float alpha = 1.0;
	float beta = 0.0;
	stat = hipblasSgemv(blasHandle, HIPBLAS_OP_N, HEIGHT, WIDTH, &alpha, blasMatrix, HEIGHT, blasVector, 1, &beta, output, 1);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		CLEANUP("cuBLAS matrix vector multiplication failed.", 1);
	}
	err1 = hipMemcpy(h_Out, output, sizeof(float) * HEIGHT, hipMemcpyDeviceToHost);
	if (err1 != hipSuccess)
	{
		CLEANUP("Failed to copy data back to host.", 1);
	}
	printf("-------------------------------------\nGPU Results:\n");		
	printf("Using BLAS:\n");
	for (int j = 0; j < HEIGHT; j++)
	{
		printf("Output: %f\n", h_Out[j]);
	}
	cudppCreate(&handle);
	CUDPPConfiguration config;
	config.algorithm = CUDPP_SORT_RADIX;
	config.datatype = CUDPP_FLOAT;
	config.options = CUDPP_OPTION_KEY_VALUE_PAIRS;

	CUDPPResult res = cudppPlan(handle, &scanplan, config, WIDTH, 1, 0);

	if (res != CUDPP_SUCCESS)
	{
		CLEANUP("Failed to create plan.", 1);
	}

	res = cudppRadixSort(scanplan, data, indices, WIDTH);
	
	if (res != CUDPP_SUCCESS)
	{
		CLEANUP("Failed to execute radix sort.", 1);
	}	
	float* initLow = **lowestAddr;
	hipDeviceSynchronize();
	findLowest<<<2, WIDTH / 4>>>(*lowestAddr, data, 0);
	hipDeviceSynchronize();
	if (**lowestAddr == initLow)
	{
		findLowest<<<2, WIDTH / 4>>>(*lowestAddr, data, 1);
		hipDeviceSynchronize();
	}
	if (**lowestAddr == initLow)
	{
		CLEANUP("Could not find zero boundary.", 1);
	}
	startAddr = **lowestAddr;
	endAddr = data + WIDTH;
		
	// perform dense sparse matrix vector multiplication, using the above sparse vector
	denseSparseMVM<<<endAddr - startAddr, HEIGHT>>>(matrix, startAddr, indices + (startAddr - data), HEIGHT, WIDTH, P);
	// reduce psums into result
	CUDPPConfiguration configReduce;
	configReduce.algorithm = CUDPP_REDUCE;
	configReduce.op = CUDPP_ADD;
	configReduce.datatype = CUDPP_FLOAT;

	res = cudppPlan(handle, &scanplan, configReduce, WIDTH, 1, 0);
	if (res != CUDPP_SUCCESS)
	{
		CLEANUP("Failed to create plan.", 1);
	}
	hipDeviceSynchronize();
	for (int i = 0; i < HEIGHT; i++)
	{
		res = cudppReduce(scanplan, output + i, P + i * WIDTH, (endAddr - startAddr)); 
		if (res != CUDPP_SUCCESS)
		{
			CLEANUP("Failed to reduce psums.", 1);
		}
	}
	
	hipDeviceSynchronize();
	err1 = hipMemcpy(h_Out, output, sizeof(float) * HEIGHT, hipMemcpyDeviceToHost);
	if (err1 != hipSuccess)
	{
		CLEANUP("Failed to copy data back to host.", 1);
	}
	printf("Start pointer: %p End pointer: %p\n", startAddr, endAddr);
	for (int i = startAddr - data; i < endAddr - data; i++)
	{
		printf("Input: %f Index: %d\n", data[i], indices[i]);
	}
	printf("Using SST sparse\n");
	for (int j = 0; j < HEIGHT; j++)
	{
		printf("Output: %f\n", h_Out[j]);
	}
	cudppDestroyPlan(scanplan);
	CLEANUP("Program completed successfully.", 0);
}
