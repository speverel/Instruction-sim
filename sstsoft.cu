#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#define WIDTH 256
#define HEIGHT 256
#define MAX_VAL 120

#define CLEANUP(s, v) \
	do { \
		printf("%s\n", s); \
		if (h_Act)			free(h_Act); \
		if (h_Mat)			free(h_Mat); \
		if (h_Out)			free(h_Out); \
		if (blasHandle)		hipblasDestroy(blasHandle); \
		if (data)			hipFree(data); \
		if (startAddr)		hipFree(startAddr); \
		if (endAddr)		hipFree(endAddr); \
		if (*lowestAddr)	hipFree(*lowestAddr); \
		if (output)			hipFree(output); \
		if (indices)		hipFree(indices); \
		if (matrix)			hipFree(matrix); \
		if (P)				hipFree(P); \
		if (blasMatrix)		hipFree(blasMatrix); \
		if (blasVector)		hipFree(blasVector); \
		hipDeviceReset(); \
		fflush(stdout); \
	} while (0); \
	return v;

__global__ void setIndices(short* indices)
{
	unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
	indices[tid] = tid;
}

__global__ void findLowest(float** result, float* data, int bias)
{
	unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (bias == 0 || tid < blockDim.x * gridDim.x - 1) // prevents last thread from accessing off the end
	{
		if (data[2 * tid + bias] == 0 && data[2 * tid + 1 + bias] > 0) // this is the zero boundary if true
			*result = data + 2 * tid + 1 + bias;	
	}
}

// launch endAddr(a) - startAddr(a) blocks of height threads each
__global__ void denseSparseMVM(float* W, float* a, short* ind, unsigned height, unsigned width, float* P)
{
	unsigned tid = threadIdx.x;
	unsigned i = blockIdx.x;
	float* pw = W + ind[i] * height;
	P[tid * height + i] = pw[tid] * a[i];
}

__global__ void clearPsums(float* P)
{
	P[blockIdx.x * blockDim.x + threadIdx.x] = 0.0;
}

/*
__global__ void sparseSparseMVM(float* W, float** Wends, int* Windices, float* a, float* aend, int* aindices, unsigned height, float** P)
{
	extern __shared__ int* Pindices;
	unsigned tid = threadIdx.x;
	for (unsigned i = 0; i <= (aend - a); i++)
	{
		int aindex = aindices[i];
		float* colEnd = Wends[aindex];
		float* colBeg = W + aindex * height;
		if (colBeg + tid <= colEnd)
		{
			int* Wcolind = Windices + aindex * height;
			int windex = Wcolind[tid];
			float psum = colBeg[tid] * a[i];
			unsigned pI = Pindices[windex];
			P[windex][pI] = psum;
			Pindices[windex] = pI + 1;
		}
	}
}	
*/

struct abs_comp
{
	inline bool operator() (float a, float b)
	{
		return fabs(a) < fabs(b);
	}
};

void merge(int low, int mid, int high, float* idata, float* odata, short* iind, short* oind)
{
	int l1, l2, i;
	for (l1 = low, l2 = mid + 1, i = low; l1 <= mid && l2 <= high; i++)
	{   
		if (idata[l1] <= idata[l2])
		{
			oind[i] = iind[l1];
			odata[i] = idata[l1++];
		}
		else
		{
			oind[i] = iind[l2];
			odata[i] = idata[l2++];
		}
	}
	while (l1 <= mid)
	{
		oind[i] = iind[l1];
		odata[i++] = idata[l1++];
	}
	while (l2 <= high)
	{
		oind[i] = iind[l2];
		odata[i++] = idata[l2++];
	}
	for (i = low; i <= high; i++)
	{
		idata[i] = odata[i];
		iind[i] = oind[i];
	}
}

void mergesort(int low, int high, float* idata, float* odata, short* iind, short* oind)
{
	int mid;
	if (low < high)
	{
		mid = (low + high) / 2;
		mergesort(low, mid, idata, odata, iind, oind);
		mergesort(mid + 1, high, idata, odata, iind, oind);
		merge(low, mid, high, idata, odata, iind, oind);
	}
	else
		return;
}

void matMul(float* matrix, float* vector, float* output, int width, int height)
{
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			output[i] += matrix[j * height + i] * vector[j];
		}
	}
}

void printGold(float* matrix, float* vector, int width, int height, float* dataPtr)
{
	float* tempData = (float*)malloc(sizeof(float) * width);
	short* indices = (short*)malloc(sizeof(short) * width);
	short* tempIndices = (short*)malloc(sizeof(short) * width);
	float* mulResult = (float*)malloc(sizeof(float) * height);
	for (int i = 0; i < width; i++)
		indices[i] = i;
	for (int i = 0; i < height; i++)
		mulResult[i] = 0.0;
	matMul(matrix, vector, mulResult, width, height);
	float *startAddr, *endAddr;
	endAddr = dataPtr + width;
	mergesort(0, width - 1, vector, tempData, indices, tempIndices);
	int index = 0;
	while (tempData[index] == 0.0)
		index++;
	startAddr = dataPtr + index;
	printf("CPU results:\n");
	printf("Start pointer: %p End pointer: %p\n", startAddr, endAddr);
	for (int i = startAddr - dataPtr; i < endAddr - dataPtr; i++)
		printf("Input: %f Index: %d\n", tempData[i], tempIndices[i]);
	for (int i = 0; i < height; i++)
		printf("Output: %f\n", mulResult[i]);	
	free(tempData);
	free(indices);
	free(tempIndices);
	free(mulResult);
}

int main()
{
	srand(time(NULL));
	float* h_Act = (float*)malloc(sizeof(float) * WIDTH);
	float* h_Mat = (float*)malloc(sizeof(float) * WIDTH * HEIGHT);
	float* h_Out = (float*)malloc(sizeof(float) * HEIGHT);
	float temp;
	hipblasStatus_t stat, stat2;
	hipblasHandle_t blasHandle;
	int nnz = 0;
	for (int i = 0; i < WIDTH; i++) // generate random floats, which are rectified
	{
		temp = (float)rand() / (float)(RAND_MAX / MAX_VAL) - MAX_VAL / 1.4;
		if (temp > 0)
		{
			h_Act[i] = temp;
			nnz++;
		}
		else
		{
			h_Act[i] = 0;
		}
	}

	// populate matrix
	for (int i = 0; i < WIDTH; i++)
	{
		for (int j = 0; j < HEIGHT; j++)
		{
			h_Mat[i * HEIGHT + j] = (float)rand() / (float)(RAND_MAX / (MAX_VAL)) - MAX_VAL / 2;
		}
	}
	// allocate memory on GPU, and copy activation vector over
	float *data, *output, *matrix, *startAddr, *endAddr, *P, *blasMatrix, *blasVector;
	short* indices;
	float** foo;
	float*** lowestAddr = &foo;
	hipError_t err1 = hipMallocManaged(&data, sizeof(float) * WIDTH, hipMemAttachGlobal);
	hipError_t err2 = hipMallocManaged(&indices, sizeof(short) * WIDTH, hipMemAttachGlobal);
	hipError_t err3 = hipMallocManaged(&startAddr, sizeof(float), hipMemAttachGlobal);
	hipError_t err4 = hipMalloc(&output, sizeof(float) * HEIGHT);
	hipError_t err5 = hipMalloc(&matrix, sizeof(float) * HEIGHT * WIDTH);
	hipError_t err6 = hipMallocManaged(lowestAddr, sizeof(float**), hipMemAttachGlobal);
	hipError_t err7 = hipMallocManaged(&endAddr, sizeof(float), hipMemAttachGlobal);
	hipError_t err8 = hipMallocManaged(&P, sizeof(float) * HEIGHT * WIDTH);
	hipError_t err9 = hipMalloc(&blasMatrix, sizeof(float) * HEIGHT * WIDTH);
	hipError_t err10 = hipMalloc(&blasVector, sizeof(float) * WIDTH);
	if (err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess || err5 != hipSuccess || err6 != hipSuccess || err7 != hipSuccess || err8 != hipSuccess || err9 != hipSuccess || err10 != hipSuccess) 
	{
		CLEANUP("Failed to allocate memory on device.", 1);
	}
	err1 = hipMemcpy(data, h_Act, sizeof(float) * WIDTH, hipMemcpyHostToDevice);
	err3 = hipMemcpy(matrix, h_Mat, sizeof(float) * WIDTH * HEIGHT, hipMemcpyHostToDevice);
	if (err1 != hipSuccess || err3 != hipSuccess)
	{
		CLEANUP("Failed to copy memory to device.", 1);
	}

	stat = hipblasCreate(&blasHandle);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		CLEANUP("Failed to create cuBLAS handle.", 1);
	}
	stat = hipblasSetMatrix(HEIGHT, WIDTH, sizeof(float), h_Mat, HEIGHT, blasMatrix, HEIGHT);
	stat2 = hipblasSetVector(WIDTH, sizeof(float), h_Act, 1, blasVector, 1);
	if (stat != HIPBLAS_STATUS_SUCCESS || stat2 != HIPBLAS_STATUS_SUCCESS)
	{
		CLEANUP("Failed to copy BLAS matrix or vector over.", 1);
	}
	printGold(h_Mat, h_Act, WIDTH, HEIGHT, data);

	setIndices<<<4, WIDTH / 4>>>(indices);
	clearPsums<<<HEIGHT, WIDTH>>>(P);

	float alpha = 1.0;
	float beta = 0.0;
	stat = hipblasSgemv(blasHandle, HIPBLAS_OP_N, HEIGHT, WIDTH, &alpha, blasMatrix, HEIGHT, blasVector, 1, &beta, output, 1);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		CLEANUP("cuBLAS matrix vector multiplication failed.", 1);
	}
	err1 = hipMemcpy(h_Out, output, sizeof(float) * HEIGHT, hipMemcpyDeviceToHost);
	if (err1 != hipSuccess)
	{
		CLEANUP("Failed to copy data back to host.", 1);
	}
	printf("-------------------------------------\nGPU Results:\n");		
	printf("Using BLAS:\n");
	for (int j = 0; j < HEIGHT; j++)
	{
		printf("Output: %f\n", h_Out[j]);
	}
	thrust::sort_by_key(data, data + WIDTH, indices, abs_comp());
	float* initLow = **lowestAddr;
	findLowest<<<2, WIDTH / 4>>>(*lowestAddr, data, 0);
	hipDeviceSynchronize();
	if (**lowestAddr == initLow)
	{
		findLowest<<<2, WIDTH / 4>>>(*lowestAddr, data, 1);
		hipDeviceSynchronize();
	}
	if (**lowestAddr == initLow)
	{
		CLEANUP("Could not find zero boundary.", 1);
	}
	startAddr = **lowestAddr;
	endAddr = data + WIDTH;
		
	// perform dense sparse matrix vector multiplication, using the above sparse vector
	denseSparseMVM<<<endAddr - startAddr, HEIGHT>>>(matrix, startAddr, indices + (startAddr - data), HEIGHT, WIDTH, P);
	// reduce psums into result
	hipDeviceSynchronize();
	for (int i = 0; i < HEIGHT; i++)
	{
		h_Out[i] = thrust::reduce(P + i * WIDTH, P + i * WIDTH + (endAddr - startAddr));
	}
	printf("Start pointer: %p End pointer: %p\n", startAddr, endAddr);
	for (int i = startAddr - data; i < endAddr - data; i++)
	{
		printf("Input: %f Index: %d\n", data[i], indices[i]);
	}
	printf("Using SST sparse\n");
	for (int j = 0; j < HEIGHT; j++)
	{
		printf("Output: %f\n", h_Out[j]);
	}
	CLEANUP("Program completed successfully.", 0);
}
